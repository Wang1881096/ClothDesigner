#include "GpuSim.h"

#include "ldpMat\ldp_basic_mat.h"
#include "cudpp\thrust_wrapper.h"
#include "cudpp\CachedDeviceBuffer.h"
namespace ldp
{
	template<class T>
	static hipTextureObject_t createTexture(DeviceArray2D<T>& ary, hipTextureFilterMode filterMode)
	{
		hipResourceDesc texRes;
		memset(&texRes, 0, sizeof(hipResourceDesc));
		texRes.resType = hipResourceTypePitch2D;
		texRes.res.pitch2D.height = ary.rows();
		texRes.res.pitch2D.width = ary.cols();
		texRes.res.pitch2D.pitchInBytes = ary.step();
		texRes.res.pitch2D.desc = hipCreateChannelDesc<T>();
		texRes.res.pitch2D.devPtr = ary.ptr();
		hipTextureDesc texDescr;
		memset(&texDescr, 0, sizeof(hipTextureDesc));
		texDescr.normalizedCoords = 0;
		texDescr.filterMode = filterMode;
		texDescr.addressMode[0] = hipAddressModeClamp;
		texDescr.addressMode[1] = hipAddressModeClamp;
		texDescr.addressMode[2] = hipAddressModeClamp;
		texDescr.readMode = hipReadModeElementType;
		hipTextureObject_t tex;
		cudaSafeCall(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));
		return tex;
	}

}