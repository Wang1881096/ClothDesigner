#include "hip/hip_runtime.h"
#include "clothManager.h"
#include "LevelSet3D.h"
#include "LEVEL_SET_COLLISION.h"
#include "cudpp\cuda_utils.h"
#include "cudpp\hip/hip_vector_types.h"

//#include "MY_MATH.h"
//#include "COLLISION_HANDLER.h"
#include "SelfCollider.h"

namespace ldp
{
	enum
	{
		threadsPerBlock = 256
	};

	__constant__ float g_gravity[3] = { 0, -9.8, 0 };

#pragma region --laplacian damping
	__global__ void Laplacian_Damping_Kernel(const float* V, float* next_V, const float* fixed, 
		const float* more_fixed, const int* all_VV, const int* all_vv_num, const int number, const float rate)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float3 nvi = make_float3(0, 0, 0);
		float3 vi = make_float3(V[i * 3 + 0], V[i * 3 + 1], V[i * 3 + 2]);
		float r = (more_fixed[i] == 0) * (fixed[i] == 0);
		const int ed = all_vv_num[i + 1];
		for (int index = all_vv_num[i]; index<ed; index++)
		{
			int j = all_VV[index];
			float3 vj = make_float3(V[j * 3 + 0], V[j * 3 + 1], V[j * 3 + 2]);
			nvi += (vj - vi) * r;
		}
		nvi = (vi + nvi * rate) * r;
		next_V[i * 3 + 0] = nvi.x;
		next_V[i * 3 + 1] = nvi.y;
		next_V[i * 3 + 2] = nvi.z;

	}
	void ClothManager::laplaceDamping()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		for (int l = 0; l<m_simulationParam.lap_damping; l++)
		{
			Laplacian_Damping_Kernel << <blocksPerGrid, threadsPerBlock >> >(
				m_dev_V.ptr(), m_dev_next_X.ptr(), m_dev_fixed.ptr(), 
				m_dev_more_fixed.ptr(), m_dev_all_VV.ptr(), m_dev_all_vv_num.ptr(), 
				m_X.size(), 0.1);
			m_dev_next_X.copyTo(m_dev_V);
		}
	}
#pragma endregion

#pragma region --update
	__global__ void Update_Kernel(float* X, float* V, float* fixed, const float* more_fixed, 
		const float damping, const float t, const int number, const float dir_x, 
		const float dir_y, const float dir_z)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		if (more_fixed[i] != 0)
		{
			X[i * 3 + 0] += dir_x;
			X[i * 3 + 1] += dir_y;
			X[i * 3 + 2] += dir_z;
			V[i * 3 + 0] = 0;
			V[i * 3 + 1] = 0;
			V[i * 3 + 2] = 0;
			return;
		}

		if (fixed[i] != 0)
		{
			V[i * 3 + 0] = 0;
			V[i * 3 + 1] = 0;
			V[i * 3 + 2] = 0;
			return;
		}

		//Apply damping
		V[i * 3 + 0] *= damping;
		V[i * 3 + 1] *= damping;
		V[i * 3 + 2] *= damping;

		//Apply gravity
		V[i * 3 + 0] += g_gravity[0] * t;
		V[i * 3 + 1] += g_gravity[1] * t;
		V[i * 3 + 2] += g_gravity[2] * t;

		//Position update
		X[i * 3 + 0] = X[i * 3 + 0] + V[i * 3 + 0] * t;
		X[i * 3 + 1] = X[i * 3 + 1] + V[i * 3 + 1] * t;
		X[i * 3 + 2] = X[i * 3 + 2] + V[i * 3 + 2] * t;
	}

	void ClothManager::updateAfterLap()
	{
		// ldp hack here: make the gravity not important when we are stitching.
		Float3 gravity = m_simulationParam.gravity * powf(1 - std::max(0.f, std::min(1.f, m_curStitchRatio)), 2);

		hipMemcpyToSymbol(HIP_SYMBOL(g_gravity), gravity.ptr(), 3 * sizeof(float));
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Update_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_V.ptr(), m_dev_fixed.ptr(), m_dev_more_fixed.ptr(), 
			m_simulationParam.air_damping, m_simulationParam.time_step, m_X.size(), 
			m_curDragInfo.dir[0], m_curDragInfo.dir[1], m_curDragInfo.dir[2]);
	}
#pragma endregion

#pragma region --constrain0
	__global__ void Constraint_0_Kernel(const float* X, float* init_B, float* new_VC, 
		const float *fixed, const float* more_fixed, const float inv_t, const int number)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float c = (1 + fixed[i] + more_fixed[i])*inv_t*inv_t;
		init_B[i * 3 + 0] = c*X[i * 3 + 0];
		init_B[i * 3 + 1] = c*X[i * 3 + 1];
		init_B[i * 3 + 2] = c*X[i * 3 + 2];
		new_VC[i] += c;
	}

	void ClothManager::constrain0()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		m_dev_all_VC.copyTo(m_dev_new_VC);
		Constraint_0_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_init_B.ptr(), m_dev_new_VC.ptr(), m_dev_fixed.ptr(), 
			m_dev_more_fixed.ptr(), 1 / m_simulationParam.time_step, m_X.size());
		cudaSafeCall(hipGetLastError(), "constrain0");
		m_dev_X.copyTo(m_dev_prev_X);
	}
#pragma endregion
	
#pragma region --constrain1
__global__ void Constraint_1_Kernel(const float* X, const float* init_B,
		float* next_X, const int* all_VV, const float* all_VL, const float* all_VW, 
		const float* new_VC, const int* all_vv_num, const float spring_k, const int number,
		const int* stitch_VV, const float* stitch_VW, const float* stitch_VC, const int* stitch_vv_num,
		const float* stitch_VL, float stitch_k, float stitch_ratio)
	{
		const int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float3 b = make_float3(init_B[i * 3 + 0], init_B[i * 3 + 1], init_B[i * 3 + 2]);
		float3 k = make_float3(0, 0, 0);
		const float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
		float diag = new_VC[i];

		const int bg = all_vv_num[i], ed = all_vv_num[i + 1];
		for (int index = bg; index<ed; index++)
		{
			const int j = all_VV[index];
			const float jl = all_VL[index];
			const float3 xj = make_float3(X[j * 3 + 0], X[j * 3 + 1], X[j * 3 + 2]);

			// Remove the off-diagonal (Jacobi method)
			b -= all_VW[index] * xj;

			// Add the other part of b: spring-length constraint
			const float3 d = normalize(xi - xj);
			b += d * spring_k* jl;
			k += (d * d + max(0., 1.-jl) * (1 - d * d) - 1) * spring_k; // ldp: what is this? cannot understand
		}

		// handel stitch
		if (stitch_VV)
		{
			const int bg = stitch_vv_num[i], ed = stitch_vv_num[i + 1];
			float sumStitchW = 0;
			const float bend_stitch_w = powf(1 - stitch_ratio, 10.f);
			for (int index = bg; index<ed; index++)
			{
				const int j = stitch_VV[index];
				const float jl = stitch_VL[index];
				const float3 xj = make_float3(X[j * 3 + 0], X[j * 3 + 1], X[j * 3 + 2]);

				// Remove the off-diagonal (Jacobi method)
				b += (jl != 0)*stitch_k*xj - bend_stitch_w*stitch_VW[index] * xj;;
				sumStitchW += (jl != 0)*stitch_k;

				// Add the other part of b: spring-length constraint
				float3 d = (xi - xj) / (length(xi - xj) + 1e-16);
				b += d * stitch_k* jl * stitch_ratio;
				k += (d * d + max(0., 1. - jl * stitch_ratio) * (1 - d * d) - 1) * stitch_k; // ldp: what is this? cannot understand
			}
			diag += sumStitchW + bend_stitch_w*stitch_VC[i];
		} // end if stitch_VV

		const float3 nxi = xi + (b - diag * xi) / (diag + k);

		next_X[i * 3 + 0] = nxi.x;
		next_X[i * 3 + 1] = nxi.y;
		next_X[i * 3 + 2] = nxi.z;
	}

	void ClothManager::constrain1()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_1_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_init_B.ptr(), m_dev_next_X.ptr(), 
			m_dev_all_VV.ptr(), m_dev_all_VL.ptr(), m_dev_all_VW.ptr(), m_dev_new_VC.ptr(), 
			m_dev_all_vv_num.ptr(), m_simulationParam.spring_k, m_X.size(),
			m_dev_stitch_VV.ptr(), m_dev_stitch_VW.ptr(), m_dev_stitch_VC.ptr(), m_dev_stitch_VV_num.ptr(),
			m_dev_stitch_VL.ptr(), m_simulationParam.stitch_k, m_curStitchRatio);
		cudaSafeCall(hipGetLastError(), "constrain1");
	}
#pragma endregion

#pragma region --constrain2
	__global__ void Constraint_2_Kernel(const float* prev_X, const float* X, 
		float* next_X, float omega, int number, float under_relax)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
		float3 nxi = make_float3(next_X[i * 3 + 0], next_X[i * 3 + 1], next_X[i * 3 + 2]);
		float3 pxi = make_float3(prev_X[i * 3 + 0], prev_X[i * 3 + 1], prev_X[i * 3 + 2]);
		nxi = (nxi - xi) * under_relax + xi;
		nxi = omega * (nxi - pxi) + pxi;
		next_X[i * 3 + 0] = nxi.x;
		next_X[i * 3 + 1] = nxi.y;
		next_X[i * 3 + 2] = nxi.z;
	}

	void ClothManager::constrain2(float omega)
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_2_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_prev_X.ptr(), m_dev_X.ptr(), m_dev_next_X.ptr(),
			omega, m_X.size(), m_simulationParam.under_relax);
		cudaSafeCall(hipGetLastError(), "constrain2");
	}
#pragma endregion

#pragma region --constrain3
	__global__ void Constraint_3_Kernel(float* X, const float *old_X, int number,
		const float *phi, const float* outgo_dist, const float3 start, 
		const float h, const float inv_h, const int size_x, const int size_y, 
		const int size_z)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	
			return;

		if (phi)
		{
			const float level_set_goal = 1.0f + outgo_dist[i] * inv_h;
			float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
			float3 oxi = make_float3(old_X[i * 3 + 0], old_X[i * 3 + 1], old_X[i * 3 + 2]);
			float3 t = (xi - start) * inv_h;
			float depth = Level_Set_Depth(phi, t.x, t.y, t.z, level_set_goal, size_x, size_y, size_z, size_y*size_z)*h;
			if (depth<0)
			{
				t = xi - oxi;
				float t_length = length(t);
				if (t_length>1e-16f)
				{
					t /= t_length;
					t_length = t_length - fabsf(depth)*1.2;
					if (t_length<0)	t_length = 0;
				}
				t = (oxi + t*t_length - start)*inv_h;
				Level_Set_Projection(phi, t.x, t.y, t.z, level_set_goal, size_x, size_y, size_z, size_y*size_z);
				X[i * 3 + 0] = t.x*h + start.x;
				X[i * 3 + 1] = t.y*h + start.y;
				X[i * 3 + 2] = t.z*h + start.z;
			} // end if depth
		} // end if phi
	}

	void ClothManager::constrain3()
	{
		const auto start = m_bodyLvSet->getStartPos();
		const auto h = m_bodyLvSet->getStep();
		const auto inv_h = 1 / h;
		const auto size = m_bodyLvSet->size();
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_3_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_old_X.ptr(), m_X.size(),
			m_dev_phi.ptr(), m_dev_V_outgo_dist.ptr(), make_float3(start[0], start[1], start[2]), h, inv_h, 
			size[0], size[1], size[2]);
		cudaSafeCall(hipGetLastError(), "constrain3");
	}
#pragma endregion

#pragma region --constrain4
	__global__ void Constraint_4_Kernel(float* X, float* init_B, float* V, const float* fixed, 
		const float* more_fixed, float inv_t, int number)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float c = (1 + fixed[i] + more_fixed[i])*inv_t*inv_t;
		V[i * 3 + 0] += (X[i * 3 + 0] - init_B[i * 3 + 0] / c)*inv_t;
		V[i * 3 + 1] += (X[i * 3 + 1] - init_B[i * 3 + 1] / c)*inv_t;
		V[i * 3 + 2] += (X[i * 3 + 2] - init_B[i * 3 + 2] / c)*inv_t;
	}


	void ClothManager::constrain4()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_4_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_init_B.ptr(), m_dev_V.ptr(), m_dev_fixed.ptr(), 
			m_dev_more_fixed.ptr(), 1 / m_simulationParam.time_step, m_X.size());
		cudaSafeCall(hipGetLastError(), "constrain4");
	}
#pragma endregion

#pragma region --drag control
#define RADIUS_SQUARED	0.000625
	__global__ void Control_Kernel(float* X, float *more_fixed, float control_mag, const int number, 
		const int select_v, const int start_id, const int end_id)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= end_id || i < start_id)	return;

		more_fixed[i] = 0;
		if (select_v != -1)
		{
			float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
			float3 xs = make_float3(X[select_v * 3 + 0], X[select_v * 3 + 1], X[select_v * 3 + 2]);
			if (dot(xi-xs, xi-xs) < RADIUS_SQUARED)	
				more_fixed[i] = control_mag;
		}
	}

	void ClothManager::resetMoreFixed()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Control_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_more_fixed.ptr(), m_simulationParam.control_mag, 
			m_X.size(), m_curDragInfo.vert_id, m_curDragInfo.piece_id_start,
			m_curDragInfo.piece_id_end);
		cudaSafeCall(hipGetLastError(), "resetMoreFixed");
	}
#pragma endregion

#pragma region --collision handler
	//std::shared_ptr<COLLISION_HANDLER> g_chl;
	void ClothManager::initCollisionHandler()
	{
		//g_chl.reset(new COLLISION_HANDLER());
		m_collider.reset(new SelfCollider());
	}

	void ClothManager::constrain_selfCollision()
	{
		if (m_simulationParam.enable_self_collistion)
		{
			m_collider->run((float3*)m_dev_old_X.ptr(), (float3*)m_dev_X.ptr(), (float3*)m_dev_V.ptr(), m_X.size(), 
				(const int3*)m_dev_T.ptr(), m_T.size(), (const float3*)m_X.data(), 1.f / m_simulationParam.time_step,
				m_dev_stitchPair_num.ptr(), m_dev_stitchPair.ptr(), m_dev_stitchPair.size());
			//g_chl->Run(m_dev_old_X.ptr(), m_dev_X.ptr(), m_dev_V.ptr(), m_X.size(),
			//	m_dev_T.ptr(), m_T.size(), (float*)m_X.data(), 1.f / m_simulationParam.time_step);
		}
	}
#pragma endregion
}