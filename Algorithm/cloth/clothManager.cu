#include "hip/hip_runtime.h"
#include "clothManager.h"
#include "cuda_utils.h"
#include "LevelSet3D.h"
#include "LEVEL_SET_COLLISION.h"
#include "hip/hip_vector_types.h"
namespace ldp
{
	enum
	{
		threadsPerBlock = 256
	};

	__constant__ float g_gravity[3] = { 0, -9.8, 0 };

#pragma region --laplacian damping
	__global__ void Laplacian_Damping_Kernel(const float* V, float* next_V, const float* fixed, 
		const float* more_fixed, const int* all_VV, const int* all_vv_num, const int number, const float rate)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float3 nvi = make_float3(0, 0, 0);
		float3 vi = make_float3(V[i * 3 + 0], V[i * 3 + 1], V[i * 3 + 2]);
		float r = (more_fixed[i] == 0) * (fixed[i] == 0);
		const int ed = all_vv_num[i + 1];
		for (int index = all_vv_num[i]; index<ed; index++)
		{
			int j = all_VV[index];
			float3 vj = make_float3(V[j * 3 + 0], V[j * 3 + 1], V[j * 3 + 2]);
			nvi += (vj - vi) * r;
		}
		nvi = (vi + nvi * rate) * r;
		next_V[i * 3 + 0] = nvi.x;
		next_V[i * 3 + 1] = nvi.y;
		next_V[i * 3 + 2] = nvi.z;

	}
	void ClothManager::laplaceDamping()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		for (int l = 0; l<m_simulationParam.lap_damping; l++)
		{
			Laplacian_Damping_Kernel << <blocksPerGrid, threadsPerBlock >> >(
				m_dev_V.ptr(), m_dev_next_X.ptr(), m_dev_fixed.ptr(), 
				m_dev_more_fixed.ptr(), m_dev_all_VV.ptr(), m_dev_all_vv_num.ptr(), 
				m_X.size(), 0.1);
			m_dev_next_X.copyTo(m_dev_V);
		}
	}
#pragma endregion

#pragma region --update
	__global__ void Update_Kernel(float* X, float* V, float* fixed, const float* more_fixed, 
		const float damping, const float t, const int number, const float dir_x, 
		const float dir_y, const float dir_z)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		if (more_fixed[i] != 0)
		{
			X[i * 3 + 0] += dir_x;
			X[i * 3 + 1] += dir_y;
			X[i * 3 + 2] += dir_z;
			V[i * 3 + 0] = 0;
			V[i * 3 + 1] = 0;
			V[i * 3 + 2] = 0;
			return;
		}

		if (fixed[i] != 0)
		{
			V[i * 3 + 0] = 0;
			V[i * 3 + 1] = 0;
			V[i * 3 + 2] = 0;
			return;
		}

		//Apply damping
		V[i * 3 + 0] *= damping;
		V[i * 3 + 1] *= damping;
		V[i * 3 + 2] *= damping;

		//Apply gravity
		V[i * 3 + 0] += g_gravity[0] * t;
		V[i * 3 + 1] += g_gravity[1] * t;
		V[i * 3 + 2] += g_gravity[2] * t;

		//Position update
		X[i * 3 + 0] = X[i * 3 + 0] + V[i * 3 + 0] * t;
		X[i * 3 + 1] = X[i * 3 + 1] + V[i * 3 + 1] * t;
		X[i * 3 + 2] = X[i * 3 + 2] + V[i * 3 + 2] * t;
	}

	void ClothManager::updateAfterLap()
	{
		hipMemcpyToSymbol(HIP_SYMBOL(g_gravity), m_simulationParam.gravity.ptr(), 3 * sizeof(float));
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Update_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_V.ptr(), m_dev_fixed.ptr(), m_dev_more_fixed.ptr(), 
			m_simulationParam.air_damping, m_simulationParam.time_step, m_X.size(), 
			m_curDragInfo.dir[0], m_curDragInfo.dir[1], m_curDragInfo.dir[2]);
	}
#pragma endregion

#pragma region --constrain0
	__global__ void Constraint_0_Kernel(const float* X, float* init_B, float* new_VC, 
		const float *fixed, const float* more_fixed, const float inv_t, const int number)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float c = (1 + fixed[i] + more_fixed[i])*inv_t*inv_t;
		init_B[i * 3 + 0] = c*X[i * 3 + 0];
		init_B[i * 3 + 1] = c*X[i * 3 + 1];
		init_B[i * 3 + 2] = c*X[i * 3 + 2];
		new_VC[i] += c;
	}

	void ClothManager::constrain0()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		m_dev_all_VC.copyTo(m_dev_new_VC);
		Constraint_0_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_init_B.ptr(), m_dev_new_VC.ptr(), m_dev_fixed.ptr(), 
			m_dev_more_fixed.ptr(), 1 / m_simulationParam.time_step, m_X.size());
		cudaSafeCall(hipGetLastError(), "constrain0");
		m_dev_X.copyTo(m_dev_prev_X);
	}
#pragma endregion
	
#pragma region --constrain1

#ifdef ENABLE_EDGE_WISE_STITCH
	__global__ void Compute_Stitch_Vec_Kernel(
		float* stitch_E_curVec, const float* X, int e_number,
		const float* stitch_EV_W, const int* stitch_EV, const int* stitch_EV_num,
		const float* stitch_E_length, const float stitch_ratio, const float stitch_k)
	{
		const int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= e_number || stitch_EV == nullptr)	return;
		const int bg = stitch_EV_num[i], ed = stitch_EV_num[i + 1];
		float3 vec = make_float3(0, 0, 0);
		for (int index = bg; index < ed; index++)
		{
			const int j = stitch_EV[index];
			const float wj = stitch_EV_W[index];
			const float3 xj = make_float3(X[j * 3 + 0], X[j * 3 + 1], X[j * 3 + 2]);
			vec += wj * xj;
		}
		vec /= length(vec) + 1e-16;
		vec *= stitch_ratio * stitch_k * stitch_E_length[i];
		stitch_E_curVec[i * 3 + 0] = vec.x;
		stitch_E_curVec[i * 3 + 1] = vec.y;
		stitch_E_curVec[i * 3 + 2] = vec.z;
	}

	__global__ void Constraint_1_Kernel_e(const float* X, const float* init_B,
		float* next_X, const int* all_VV, const float* all_VL, const float* all_VW,
		const float* new_VC, const int* all_vv_num, const float spring_k, const int number,
		const int* stitch_VV, const float* stitch_VW, const float* stitch_VC, const int* stitch_vv_num,
		const int* stitch_VE, const int* stitch_VE_num, const float* stitch_VE_W, 
		const float* stitch_E_curVec)
	{
		const int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float3 b = make_float3(init_B[i * 3 + 0], init_B[i * 3 + 1], init_B[i * 3 + 2]);
		float3 k = make_float3(0, 0, 0);
		const float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
		float diag = new_VC[i];

		const int bg = all_vv_num[i], ed = all_vv_num[i + 1];
		for (int index = bg; index<ed; index++)
		{
			const int j = all_VV[index];
			const float3 xj = make_float3(X[j * 3 + 0], X[j * 3 + 1], X[j * 3 + 2]);
			const float3 d = normalize(xi - xj);

			// Remove the off-diagonal (Jacobi method)
			b -= all_VW[index] * xj;

			// Add the other part of b: spring-length constraint
			b += d * spring_k * all_VL[index];
		}

		// handel stitch
		if (stitch_VV)
		{	
			const int bg = stitch_vv_num[i], ed = stitch_vv_num[i + 1];
			for (int index = bg; index < ed; index++)
			{
				const int j = stitch_VV[index];
				const float3 xj = make_float3(X[j * 3 + 0], X[j * 3 + 1], X[j * 3 + 2]);
				b -= stitch_VW[index] * xj;
			}
			diag += stitch_VC[i];

			const int bg1 = stitch_VE_num[i], ed1 = stitch_VE_num[i + 1];
			for (int index = bg1; index < ed1; index++)
			{
				const int j = stitch_VE[index];
				const float3 d = make_float3(stitch_E_curVec[j * 3 + 0], 
					stitch_E_curVec[j * 3 + 1], stitch_E_curVec[j * 3 + 2]);
				b += stitch_VE_W[index] * d;
			}
		} // end if stitch_VV

		const float3 nxi = xi + (b - diag * xi) / (diag + k);

		next_X[i * 3 + 0] = nxi.x;
		next_X[i * 3 + 1] = nxi.y;
		next_X[i * 3 + 2] = nxi.z;
	}

	void ClothManager::constrain1()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Compute_Stitch_Vec_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_stitchE_curVec.ptr(), m_dev_X.ptr(), m_stitchE_length.size(),
			m_dev_stitchEV_W.ptr(), m_dev_stitchEV.ptr(), m_dev_stitchEV_num.ptr(),
			m_dev_stitchE_length.ptr(), m_curStitchRatio, m_simulationParam.stitch_k
			);
		Constraint_1_Kernel_e << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_init_B.ptr(), m_dev_next_X.ptr(),
			m_dev_all_VV.ptr(), m_dev_all_VL.ptr(), m_dev_all_VW.ptr(), m_dev_new_VC.ptr(),
			m_dev_all_vv_num.ptr(), m_simulationParam.spring_k, m_X.size(),
			m_dev_stitch_VV.ptr(), m_dev_stitch_VW.ptr(), m_dev_stitch_VC.ptr(), m_dev_stitch_VV_num.ptr(),
			m_dev_stitchVE.ptr(), m_dev_stitchVE_num.ptr(), m_dev_stitchVE_W.ptr(), m_dev_stitchE_curVec.ptr());
		cudaSafeCall(hipGetLastError(), "constrain1");
	}
#else
__global__ void Constraint_1_Kernel(const float* X, const float* init_B,
		float* next_X, const int* all_VV, const float* all_VL, const float* all_VW, 
		const float* new_VC, const int* all_vv_num, const float spring_k, const int number,
		const int* stitch_VV, const float* stitch_VW, const float* stitch_VC, const int* stitch_vv_num,
		const float* stitch_VL, float stitch_k, float stitch_ratio)
	{
		const int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float3 b = make_float3(init_B[i * 3 + 0], init_B[i * 3 + 1], init_B[i * 3 + 2]);
		float3 k = make_float3(0, 0, 0);
		const float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
		float diag = new_VC[i];

		const int bg = all_vv_num[i], ed = all_vv_num[i + 1];
		for (int index = bg; index<ed; index++)
		{
			const int j = all_VV[index];
			const float jl = all_VL[index];
			const float3 xj = make_float3(X[j * 3 + 0], X[j * 3 + 1], X[j * 3 + 2]);

			// Remove the off-diagonal (Jacobi method)
			b -= all_VW[index] * xj;

			// Add the other part of b: spring-length constraint
			const float3 d = normalize(xi - xj);
			b += d * spring_k* jl;
			k += (d * d + max(0., 1.-jl) * (1 - d * d) - 1) * spring_k; // ldp: what is this? cannot understand
		}

		// handel stitch
		if (stitch_VV)
		{
			const int bg = stitch_vv_num[i], ed = stitch_vv_num[i + 1];
			float sumStitchW = 0;
			const float bend_stitch_w = powf(1 - stitch_ratio, 10.f);
			for (int index = bg; index<ed; index++)
			{
				const int j = stitch_VV[index];
				const float jl = stitch_VL[index];
				const float3 xj = make_float3(X[j * 3 + 0], X[j * 3 + 1], X[j * 3 + 2]);

				// Remove the off-diagonal (Jacobi method)
				b += (jl != 0)*stitch_k*xj - bend_stitch_w*stitch_VW[index] * xj;;
				sumStitchW += (jl != 0)*stitch_k;

				// Add the other part of b: spring-length constraint
				float3 d = (xi - xj) / (length(xi - xj) + 1e-16);
				b += d * stitch_k* jl * stitch_ratio;
				k += (d * d + max(0., 1. - jl * stitch_ratio) * (1 - d * d) - 1) * stitch_k; // ldp: what is this? cannot understand
			}
			diag += sumStitchW + bend_stitch_w*stitch_VC[i];
		} // end if stitch_VV

		const float3 nxi = xi + (b - diag * xi) / (diag + k);

		next_X[i * 3 + 0] = nxi.x;
		next_X[i * 3 + 1] = nxi.y;
		next_X[i * 3 + 2] = nxi.z;
	}

	void ClothManager::constrain1()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_1_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_init_B.ptr(), m_dev_next_X.ptr(), 
			m_dev_all_VV.ptr(), m_dev_all_VL.ptr(), m_dev_all_VW.ptr(), m_dev_new_VC.ptr(), 
			m_dev_all_vv_num.ptr(), m_simulationParam.spring_k, m_X.size(),
			m_dev_stitch_VV.ptr(), m_dev_stitch_VW.ptr(), m_dev_stitch_VC.ptr(), m_dev_stitch_VV_num.ptr(),
			m_dev_stitch_VL.ptr(), m_simulationParam.stitch_k, m_curStitchRatio);
		cudaSafeCall(hipGetLastError(), "constrain1");
	}
#endif
#pragma endregion

#pragma region --constrain2
	__global__ void Constraint_2_Kernel(const float* prev_X, const float* X, 
		float* next_X, float omega, int number, float under_relax)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
		float3 nxi = make_float3(next_X[i * 3 + 0], next_X[i * 3 + 1], next_X[i * 3 + 2]);
		float3 pxi = make_float3(prev_X[i * 3 + 0], prev_X[i * 3 + 1], prev_X[i * 3 + 2]);
		nxi = (nxi - xi) * under_relax + xi;
		nxi = omega * (nxi - pxi) + pxi;
		next_X[i * 3 + 0] = nxi.x;
		next_X[i * 3 + 1] = nxi.y;
		next_X[i * 3 + 2] = nxi.z;
	}

	void ClothManager::constrain2(float omega)
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_2_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_prev_X.ptr(), m_dev_X.ptr(), m_dev_next_X.ptr(),
			omega, m_X.size(), m_simulationParam.under_relax);
		cudaSafeCall(hipGetLastError(), "constrain2");
	}
#pragma endregion

#pragma region --constrain3
	__global__ void Constraint_3_Kernel(float* X, const float *old_X, int number,
		const float *phi, const float3 start, 
		const float h, const float inv_h, const int size_x, const int size_y, 
		const int size_z)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	
			return;

		if (phi)
		{
			float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
			float3 oxi = make_float3(old_X[i * 3 + 0], old_X[i * 3 + 1], old_X[i * 3 + 2]);
			float3 t = (xi - start) * inv_h;
			float depth = Level_Set_Depth(phi, t.x, t.y, t.z, 1.0f, size_x, size_y, size_z, size_y*size_z)*h;
			if (depth<0)
			{
				t = xi - oxi;
				float t_length = length(t);
				if (t_length>1e-16f)
				{
					t /= t_length;
					t_length = t_length - fabsf(depth)*1.2;
					if (t_length<0)	t_length = 0;
				}
				t = (oxi + t*t_length - start)*inv_h;
				Level_Set_Projection(phi, t.x, t.y, t.z, 1.0f, size_x, size_y, size_z, size_y*size_z);
				X[i * 3 + 0] = t.x*h + start.x;
				X[i * 3 + 1] = t.y*h + start.y;
				X[i * 3 + 2] = t.z*h + start.z;
			} // end if depth
		} // end if phi
	}

	void ClothManager::constrain3()
	{
		const auto start = m_bodyLvSet->getStartPos();
		const auto h = m_bodyLvSet->getStep();
		const auto inv_h = 1 / h;
		const auto size = m_bodyLvSet->size();
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_3_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_old_X.ptr(), m_X.size(),
			m_dev_phi.ptr(), make_float3(start[0], start[1], start[2]), h, inv_h, 
			size[0], size[1], size[2]);
		cudaSafeCall(hipGetLastError(), "constrain3");
	}
#pragma endregion

#pragma region --constrain4
	__global__ void Constraint_4_Kernel(float* X, float* init_B, float* V, const float* fixed, 
		const float* more_fixed, float inv_t, int number)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		float c = (1 + fixed[i] + more_fixed[i])*inv_t*inv_t;
		V[i * 3 + 0] += (X[i * 3 + 0] - init_B[i * 3 + 0] / c)*inv_t;
		V[i * 3 + 1] += (X[i * 3 + 1] - init_B[i * 3 + 1] / c)*inv_t;
		V[i * 3 + 2] += (X[i * 3 + 2] - init_B[i * 3 + 2] / c)*inv_t;
	}


	void ClothManager::constrain4()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Constraint_4_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_init_B.ptr(), m_dev_V.ptr(), m_dev_fixed.ptr(), 
			m_dev_more_fixed.ptr(), 1 / m_simulationParam.time_step, m_X.size());
		cudaSafeCall(hipGetLastError(), "constrain4");
	}
#pragma endregion

#pragma region --drag control
#define RADIUS_SQUARED	0.000625
	__global__ void Control_Kernel(float* X, float *more_fixed, float control_mag, const int number, const int select_v)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i >= number)	return;

		more_fixed[i] = 0;
		if (select_v != -1)
		{
			float3 xi = make_float3(X[i * 3 + 0], X[i * 3 + 1], X[i * 3 + 2]);
			float3 xs = make_float3(X[select_v * 3 + 0], X[select_v * 3 + 1], X[select_v * 3 + 2]);
			if (dot(xi-xs, xi-xs) < RADIUS_SQUARED)	
				more_fixed[i] = control_mag;
		}
	}

	void ClothManager::resetMoreFixed()
	{
		const int blocksPerGrid = divUp(m_X.size(), threadsPerBlock);
		Control_Kernel << <blocksPerGrid, threadsPerBlock >> >(
			m_dev_X.ptr(), m_dev_more_fixed.ptr(), m_simulationParam.control_mag, 
			m_X.size(), m_curDragInfo.vert_id);
		cudaSafeCall(hipGetLastError(), "resetMoreFixed");
	}
#pragma endregion
}